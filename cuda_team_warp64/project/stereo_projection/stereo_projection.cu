#include "hip/hip_runtime.h"
/******************************************************************************
 * Author: Shiv
 * Date: 16/03/14
 * stereo_projection.cu - (kernels with caller)
	- finds depth given two images (one shifted relative to other along x-axis)
 ******************************************************************************/

#include "stereo_projection.h"

// cuda helpers by lab instructors
#include <aux.h>
// FIX
#include <global_idx.h>
#include <global_idx.cu>


__global__ void calc_data_term(float *d_imgLeft, float *d_imgRight, float *d_g, dim3 imgDims, uint32_t nc, dim3 convexGridDims, float MU) {
    // get global idx in convex grid space
    dim3 globalIdx = globalIdx_Dim3();
    // get global idx in image plane (channels exclusive)
    dim3 globalIdx_XY = globalIdx_Dim2();

    // only threads inside convex grid space computes
    if (globalIdx.x < convexGridDims.x && globalIdx.y < convexGridDims.y && globalIdx.z < convexGridDims.z) {
		// get linear index in convex grid space
        size_t id = linearize_globalIdx(globalIdx, convexGridDims);
    	// get linear index in XY
        size_t id_XY = linearize_globalIdx(globalIdx_XY, imgDims);

        // to store calc of data term for current thread
        float g = 0.f;

        // for all channels
        for(uint32_t ch_i = 0; ch_i < nc; ch_i++) {
            // channel offset
            size_t chOffset = (size_t) imgDims.x * imgDims.y * ch_i;

            // get linear ids of shifted pixel in right image with clamping

            size_t shiftedPixel = linearize_neighbour_globalIdx(globalIdx_XY, imgDims, make_int3(globalIdx.z, 0, 0));

            // calculate difference in intensity for current channel and shift
            g += fabsf(d_imgRight[id_XY + chOffset] - (globalIdx_XY.x + globalIdx.z >= imgDims.x ? 0.f : d_imgLeft[shiftedPixel + chOffset]));
        }

        // store final calculation
        d_g[id] = MU * g;
    }
}


__global__ void update_dual(float *d_vCap, float *d_g, float *d_phiX, float *d_phiY, float *d_phiZ, dim3 convexGridDims, float SIGMA) {
    // get global idx in convex grid space
    dim3 globalIdx = globalIdx_Dim3();

    // only threads inside convex grid space computes
    if (globalIdx.x < convexGridDims.x && globalIdx.y < convexGridDims.y && globalIdx.z < convexGridDims.z) {
        // get linear index in convex grid space
        size_t id = linearize_globalIdx(globalIdx, convexGridDims);

        // get linear ids of neighbours of offset +1 in x, y and z dir
        size_t neighX = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(1, 0, 0));
        size_t neighY = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(0, 1, 0));
        size_t neighZ = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(0, 0, 1));

        // chalculate differentials along x, y and z of primal variable cap and update of dual variable without projection
        float v = d_vCap[id];
        float phiX = d_phiX[id] + SIGMA * ((globalIdx.x + 1) < convexGridDims.x ? d_vCap[neighX] - v : 0);
        float phiY = d_phiY[id] + SIGMA * ((globalIdx.y + 1) < convexGridDims.y ? d_vCap[neighY] - v : 0);
        float phiZ = d_phiZ[id] + SIGMA * ((globalIdx.z + 1) < convexGridDims.z ? d_vCap[neighZ] - v : 0);

        // update the dual variable with projection back to set K
        float3 q = make_float3(phiX, phiY, phiZ + d_g[id]);
        float trunk = fmaxf(1.f, sqrtf(powf(q.x, 2) + powf(q.y, 2)));
        float3 p = make_float3(q.x / trunk, q.y / trunk, fmaxf(0.f, q.z));
        d_phiX[id] = p.x;
        d_phiY[id] = p.y;
        d_phiZ[id] = p.z - d_g[id];
    }
}


__global__ void update_primal_and_extrapolate(float *d_vn, float *d_phiX, float *d_phiY, float *d_phiZ, float *d_vCap, dim3 convexGridDims, float TAU) {
    // get global idx in convex grid space
    dim3 globalIdx = globalIdx_Dim3();

    // only threads inside convex grid space computes
    if (globalIdx.x < convexGridDims.x && globalIdx.y < convexGridDims.y && globalIdx.z < convexGridDims.z) {
        // get linear index in convex grid space
        size_t id = linearize_globalIdx(globalIdx, convexGridDims);

        // get linear ids of neighbours of offset -1 in x, y and z dir
        size_t neighX = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(-1, 0, 0));
        size_t neighY = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(0, -1, 0));
        size_t neighZ = linearize_neighbour_globalIdx(globalIdx, convexGridDims, make_int3(0, 0, -1));

        // calculate divergence
        float dxPhiX = ((globalIdx.x + 1) < convexGridDims.x ? d_phiX[id] : 0) - (globalIdx.x > 0 ? d_phiX[neighX] : 0);
        float dyPhiY = ((globalIdx.y + 1) < convexGridDims.y ? d_phiY[id] : 0) - (globalIdx.y > 0 ? d_phiY[neighY] : 0);
        float dzPhiZ = ((globalIdx.z + 1) < convexGridDims.z ? d_phiZ[id] : 0) - (globalIdx.z > 0 ? d_phiZ[neighZ] : 0);
        float divPhi = dxPhiX + dyPhiY + dzPhiZ;

        // temporary udpate calc of vn+1 with projection back to set C with boundary conditions preserved
        float vnOld = d_vn[id];
        float vnNew = globalIdx.z == 0 ? 1.f : (float)(globalIdx.z != convexGridDims.z - 1) * fminf(1.f, fmaxf(0.f, vnOld + TAU * divPhi));

        // update to actual memory
        d_vn[id] = vnNew;
        d_vCap[id] = 2.f * vnNew - vnOld;
    }
}


__global__ void init_primal_dual(float *d_v, float *d_vCap, float *d_phiX, float *d_phiY, float *d_phiZ, dim3 convexGridDims) {
	// get global idx in convex grid space
    dim3 globalIdx = globalIdx_Dim3();

	// only threads inside convex grid space computes
    if (globalIdx.x < convexGridDims.x && globalIdx.y < convexGridDims.y && globalIdx.z < convexGridDims.z) {
    	// get linear index in convex grid space
        size_t id = linearize_globalIdx(globalIdx, convexGridDims);

        // since convex problem so doesn't matter on initialisation as long as satisfies definitions of C and K
        d_v[id] = globalIdx.z == 0 ? 1.f : 0.f;
        d_vCap[id] = 0.f;
        d_phiX[id] = 0.f;
        d_phiY[id] = 0.f;
        d_phiZ[id] = 0.f;
    }
}


__global__ void compute_depth_map(float *d_v, float *d_depthMap, dim3 convexGridDims, dim3 imgDims) {
    // get global idx in image plane (channels exclusive)
    dim3 globalIdx_XY = globalIdx_Dim2();

     // only threads inside image boundary computes
    if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y) {
    	// get cur pos in v matrix
        size_t id = linearize_globalIdx(globalIdx_XY, imgDims);

        // stride for moving along z axis
        size_t imgSize = imgDims.x * imgDims.y;

        // sum over z axis with thresholding
        float sum = 0.f, value;
        for(uint32_t z = 0; z < convexGridDims.z; z++) sum += ( (value = d_v[id + z * imgSize]) > 0.5f ? value : 0.f);

        // store calculated depth map
        d_depthMap[id] = sum;
    }
}


void stereo_projection_PD(float *h_imgLeft, float *h_imgRight, float  *h_depthMap, dim3 imgDims, uint32_t nc, dim3 convexGridDims, uint32_t steps, float MU, float SIGMA, float TAU) {
	// some sizes in bytes
    size_t imgSizeBytes = (size_t) imgDims.x * imgDims.y * nc * sizeof(float);
    size_t convexGridSizeBytes = (size_t) convexGridDims.x * convexGridDims.y * convexGridDims.z * sizeof(float);
    size_t depthMapSizeBytes = (size_t) imgDims.x * imgDims.y * sizeof(float);

    // alloc GPU memory and copy data
    float *d_imgLeft, *d_imgRight, *d_g, *d_vn, *d_vCap, *d_phiX, *d_phiY, *d_phiZ, *d_depthMap;
    hipMalloc((void **) &d_imgLeft, imgSizeBytes);
    CUDA_CHECK;
    hipMemcpy(d_imgLeft, h_imgLeft, imgSizeBytes, hipMemcpyHostToDevice);
    CUDA_CHECK;
    hipMalloc((void **) &d_imgRight, imgSizeBytes);
    CUDA_CHECK;
    hipMemcpy(d_imgRight, h_imgRight, imgSizeBytes, hipMemcpyHostToDevice);
    CUDA_CHECK;
    hipMalloc((void **) &d_g, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_vn, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_vCap, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_phiX, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_phiY, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_phiZ, convexGridSizeBytes);
    CUDA_CHECK;
    hipMalloc((void **) &d_depthMap, depthMapSizeBytes);
    CUDA_CHECK;

    // define block and grid for convex grid size
    dim3 block = dim3(8, 8, 8);
    dim3 grid = dim3((convexGridDims.x + block.x - 1) / block.x, (convexGridDims.y + block.y - 1) / block.y, (convexGridDims.z + block.z - 1) / block.z);

    //calculate data term
    calc_data_term<<<grid, block>>>(d_imgLeft, d_imgRight, d_g, imgDims, nc, convexGridDims, MU);
    // init primal dual
    init_primal_dual<<<grid, block>>>(d_vn, d_vCap, d_phiX, d_phiY, d_phiZ, convexGridDims);

    // for each time step
    for(uint32_t tStep = 0; tStep < steps; tStep++) {
    	// update dual
        update_dual<<<grid, block>>>(d_vCap, d_g, d_phiX, d_phiY, d_phiZ, convexGridDims, SIGMA);
    	// update primal and extrapolate
        update_primal_and_extrapolate<<<grid, block>>>(d_vn, d_phiX, d_phiY, d_phiZ, d_vCap, convexGridDims, TAU);
    }

    // define block and grid for computing depth map
    block = dim3(16, 16, 1);
    grid = dim3((imgDims.x + block.x - 1) / block.x, (imgDims.y + block.y - 1) / block.y, 1);

    // compute depth map
    compute_depth_map<<<grid, block>>>(d_vn, d_depthMap, convexGridDims, imgDims);

    // copy back data
    hipMemcpy(h_depthMap, d_depthMap, depthMapSizeBytes, hipMemcpyDeviceToHost);
    CUDA_CHECK;

    // free allocations
    hipFree(d_imgLeft);
    CUDA_CHECK;
    hipFree(d_imgRight);
    CUDA_CHECK;
    hipFree(d_g);
    CUDA_CHECK;
    hipFree(d_vn);
    CUDA_CHECK;
    hipFree(d_vCap);
    CUDA_CHECK;
    hipFree(d_phiX);
    CUDA_CHECK;
    hipFree(d_phiY);
    CUDA_CHECK;
    hipFree(d_phiZ);
    CUDA_CHECK;
    hipFree(d_depthMap);
    CUDA_CHECK;
}
