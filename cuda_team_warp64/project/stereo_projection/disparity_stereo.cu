#include "hip/hip_runtime.h"
/*
 * disparity_stereo.cu
 *
 *  Created on: Mar 14, 2014
 *      Author: p054
 */
#include "disparity_stereo.h"
// FIX
#include <global_idx.h>

void
disparity_computation_caller(float *h_imgInleft, float *h_imgInright,
    float *h_imgOut, dim3 imgDims, uint32_t nc, float sigma, float tau,
    uint32_t steps, uint32_t mu, uint32_t disparities)
{

  // size with channels
  size_t imgSizeBytes = (size_t) imgDims.x * imgDims.y * nc * sizeof(float);
  size_t imgOutSizeBytes = (size_t) imgDims.x * imgDims.y * sizeof(float);

  std::cout << "From disparities : " << disparities << std::endl;
  // alloc GPU memory and copy data
  float *d_imgInleft;
  float *d_imgInright;
  float *d_imgOutOld[disparities], *d_imgOutFit[disparities], *d_f,
      *d_phiX[disparities], *d_phiY[disparities], *d_phiZ[disparities];

  // Memcpy the host images
  hipMalloc((void **) &d_imgInleft, imgSizeBytes);
  CUDA_CHECK;
  hipMemcpy(d_imgInleft, h_imgInleft, imgSizeBytes, hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void **) &d_imgInright, imgSizeBytes);
  CUDA_CHECK;
  hipMemcpy(d_imgInright, h_imgInright, imgSizeBytes, hipMemcpyHostToDevice);
  CUDA_CHECK;

  // 1D long big array allocated for facilitating textures
  hipMalloc((void **) &d_f, (size_t) imgOutSizeBytes * disparities);
  CUDA_CHECK;

  for (uint32_t i = 0; i < disparities; i++)
    {
      hipMalloc((void **) &(d_imgOutOld[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_imgOutFit[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiX[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiY[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiZ[i]), imgOutSizeBytes);
      CUDA_CHECK;
    }

  //Assigning the 1d array's pointers to the cuda mem
  float ** dptr_imgOutFit, **dptr_imgOutOld, **dptr_phiX, **dptr_phiY,
      **dptr_phiZ;
  hipMalloc((void ***) &dptr_imgOutFit, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_imgOutFit, d_imgOutFit, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_imgOutOld, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_imgOutOld, d_imgOutOld, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiX, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiX, d_phiX, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiY, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiY, d_phiY, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiZ, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiZ, d_phiZ, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;

  // define block and grid
  dim3 block = dim3(16, 16, 1);
  dim3 grid = dim3((imgDims.x + block.x - 1) / block.x,
      (imgDims.y + block.y - 1) / block.y, 1);

  //Init all the terms with the dataterm
  initialize<<<grid, block>>>(d_f, d_imgInleft, d_imgInright, nc, imgDims,
      dptr_imgOutOld, dptr_imgOutFit, disparities, mu);

  initialize_dual<<<grid, block>>>(dptr_phiX, dptr_phiY, dptr_phiZ,
      disparities, imgDims);

  // for each time step
  for (uint32_t tStep = 0; tStep < steps; tStep++)
    {
      regularizer_update<<<grid, block>>>(dptr_phiX, dptr_phiY, dptr_phiZ,
          dptr_imgOutFit, d_f, sigma, imgDims, disparities);
      variational_update<<<grid, block>>>( dptr_imgOutOld,
          dptr_phiX, dptr_phiY, dptr_phiZ, dptr_imgOutFit, tau, imgDims,
          disparities);
    }

  float *d_imgOut;
  hipMalloc((void **) &d_imgOut, imgOutSizeBytes);
  CUDA_CHECK;

  layers_summation<<<grid, block>>>(d_imgOut, dptr_imgOutOld, disparities,
      imgDims);

  hipMemcpy(h_imgOut, d_imgOut, imgOutSizeBytes, hipMemcpyDeviceToHost);
  CUDA_CHECK;

// free allocations
  hipFree(d_imgInleft);
  CUDA_CHECK;
  hipFree(d_imgInright);
  CUDA_CHECK;

  for (uint32_t disparity = 0; disparity < disparities; disparity++)
    {
      hipFree(d_imgOutOld[disparity]);
      CUDA_CHECK;
      hipFree(d_imgOutFit[disparity]);
      CUDA_CHECK;
      hipFree(d_phiX[disparity]);
      CUDA_CHECK;
      hipFree(d_phiY[disparity]);
      CUDA_CHECK;
      hipFree(d_phiZ[disparity]);
      CUDA_CHECK;
    }

  hipFree(d_imgOut);
  CUDA_CHECK;
  hipFree(dptr_imgOutFit);
  CUDA_CHECK;
  hipFree(dptr_imgOutOld);
  CUDA_CHECK;
  hipFree(d_f);
  CUDA_CHECK;
  hipFree(dptr_phiX);
  CUDA_CHECK;
  hipFree(dptr_phiY);
  CUDA_CHECK;
  hipFree(dptr_phiZ);
  CUDA_CHECK;
}

void
disparity_computation_caller_tm(float *h_imgInleft, float *h_imgInright,
    float *h_imgOut, dim3 imgDims, uint32_t nc, float sigma, float tau,
    uint32_t steps, uint32_t mu, uint32_t disparities)
{
  // size with channels
  size_t imgSizeBytes = (size_t) imgDims.x * imgDims.y * nc * sizeof(float);
  size_t imgOutSizeBytes = (size_t) imgDims.x * imgDims.y * sizeof(float);

  std::cout << "From disparities : " << disparities << std::endl;
  // alloc GPU memory and copy data
  float *d_imgInleft;
  float *d_imgInright;
  float *d_imgOutOld[disparities], *d_imgOutFit[disparities], *d_f,
      *d_phiX[disparities], *d_phiY[disparities], *d_phiZ[disparities];

  // Memcpy the host images
  hipMalloc((void **) &d_imgInleft, imgSizeBytes);
  CUDA_CHECK;
  hipMemcpy(d_imgInleft, h_imgInleft, imgSizeBytes, hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void **) &d_imgInright, imgSizeBytes);
  CUDA_CHECK;
  hipMemcpy(d_imgInright, h_imgInright, imgSizeBytes, hipMemcpyHostToDevice);
  CUDA_CHECK;

  // 1D long big array allocated for facilitating textures
  hipMalloc((void **) &d_f, (size_t) imgOutSizeBytes * disparities);
  CUDA_CHECK;

  for (uint32_t i = 0; i < disparities; i++)
    {
      hipMalloc((void **) &(d_imgOutOld[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_imgOutFit[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiX[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiY[i]), imgOutSizeBytes);
      CUDA_CHECK;
      hipMalloc((void **) &(d_phiZ[i]), imgOutSizeBytes);
      CUDA_CHECK;
    }

  //Assigning the 1d array's pointers to the cuda mem
  float ** dptr_imgOutFit, **dptr_imgOutOld, **dptr_phiX, **dptr_phiY,
      **dptr_phiZ;
  hipMalloc((void ***) &dptr_imgOutFit, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_imgOutFit, d_imgOutFit, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_imgOutOld, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_imgOutOld, d_imgOutOld, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiX, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiX, d_phiX, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiY, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiY, d_phiY, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMalloc((void ***) &dptr_phiZ, sizeof(float *) * disparities);
  CUDA_CHECK;
  hipMemcpy(dptr_phiZ, d_phiZ, sizeof(float *) * disparities,
      hipMemcpyHostToDevice);
  CUDA_CHECK;

  // access as (x+0.5f,y+0.5f), not as ((x+0.5f)/w,(y+0.5f)/h)
  texRefleftImage.normalized = false;
  texRefrightImage.normalized = false;

  // no of bits for each texture channel
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  hipBindTexture2D(NULL, &texRefleftImage, d_imgInleft, &desc,
      (size_t) imgDims.x, (size_t) imgDims.y * nc,
      (size_t) imgDims.x * sizeof(d_imgInleft[0]));
  CUDA_CHECK;
  hipBindTexture2D(NULL, &texRefrightImage, d_imgInright, &desc,
      (size_t) imgDims.x, (size_t) imgDims.y * nc,
      (size_t) imgDims.x * sizeof(d_imgInright[0]));
  CUDA_CHECK;

  // define block and grid
  dim3 block = dim3(16, 16, 1);
  dim3 grid = dim3((imgDims.x + block.x - 1) / block.x,
      (imgDims.y + block.y - 1) / block.y, 1);

  //Init the dataterms and the primals
  initialize_tm<<<grid, block>>>(d_f, nc, imgDims, dptr_imgOutOld,
      dptr_imgOutFit, disparities, mu);

  initialize_dual<<<grid, block>>>(dptr_phiX, dptr_phiY, dptr_phiZ,
      disparities, imgDims);

  // Allocating virtual 3D array
  hipArray* cudaarray;
  hipExtent volumesize;
  //set cuda array volume size
  volumesize = make_hipExtent(imgDims.x, imgDims.y, disparities);
  //allocate device memory for cuda array
  hipMalloc3DArray(&cudaarray, &desc, volumesize);
  CUDA_CHECK;

  // 3D memcpy parameters
  hipMemcpy3DParms copyparms =
    { 0 };
  CUDA_CHECK;
  copyparms.extent = volumesize;
  copyparms.dstArray = cudaarray;
  copyparms.kind = hipMemcpyDefault;

  copyparms.srcPtr = make_hipPitchedPtr(d_f,
      (size_t) imgDims.x * sizeof(float), (size_t) imgDims.x,
      (size_t) imgDims.y);
  hipMemcpy3D(&copyparms);
  CUDA_CHECK;

  texRefDataTerm.filterMode = hipFilterModePoint;
  texRefDataTerm.normalized = false;
  hipBindTextureToArray(texRefDataTerm, cudaarray, desc);

  // for each time step
  for (uint32_t tStep = 0; tStep < steps; tStep++)
    {

      regularizer_update_tm<<<grid, block>>>(dptr_phiX, dptr_phiY, dptr_phiZ,
          dptr_imgOutFit, sigma, imgDims, disparities);

      variational_update<<<grid, block>>>( dptr_imgOutOld,
          dptr_phiX, dptr_phiY, dptr_phiZ, dptr_imgOutFit, tau, imgDims,
          disparities);

    }

  float *d_imgOut;
  hipMalloc((void **) &d_imgOut, imgOutSizeBytes);
  CUDA_CHECK;

  layers_summation<<<grid, block>>>(d_imgOut, dptr_imgOutOld, disparities,
      imgDims);

  hipMemcpy(h_imgOut, d_imgOut, imgOutSizeBytes, hipMemcpyDeviceToHost);
  CUDA_CHECK;

// free allocations
  hipFree(d_imgInleft);
  CUDA_CHECK;
  hipFree(d_imgInright);
  CUDA_CHECK;

  for (uint32_t disparity = 0; disparity < disparities; disparity++)
    {
      hipFree(d_imgOutOld[disparity]);
      CUDA_CHECK;
      hipFree(d_imgOutFit[disparity]);
      CUDA_CHECK;
      hipFree(d_phiX[disparity]);
      CUDA_CHECK;
      hipFree(d_phiY[disparity]);
      CUDA_CHECK;
      hipFree(d_phiZ[disparity]);
      CUDA_CHECK;
    }

  hipFree(d_imgOut);
  CUDA_CHECK;
  hipFree(dptr_imgOutFit);
  CUDA_CHECK;
  hipFree(dptr_imgOutOld);
  CUDA_CHECK;
  hipFree(d_f);
  CUDA_CHECK;
  hipFree(dptr_phiX);
  CUDA_CHECK;
  hipFree(dptr_phiY);
  CUDA_CHECK;
  hipFree(dptr_phiZ);
  CUDA_CHECK;

  hipFreeArray(cudaarray);
  hipUnbindTexture(texRefleftImage);
  hipUnbindTexture(texRefrightImage);
  hipUnbindTexture(texRefDataTerm);
}

__global__ void
initialize(float *d_f, float *d_imgInleft, float *d_imgInright, uint32_t nc,
    dim3 imgDims, float **d_imgOutOld, float **d_imgOutFit,
    uint32_t disparities, uint32_t mu)
{

  dim3 globalIdx_XY = globalIdx_Dim2();

  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      float init_value;
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          init_value = 0.f;
          // for all channels
          for (uint32_t ch_i = 0; ch_i < nc; ch_i++)
            {
              // channel offset
              size_t chOffset = (size_t) imgDims.x * imgDims.y * ch_i;
              // get linear ids of shifted pixel in right image with clamping

              init_value += fabsf(
                  d_imgInleft[id + chOffset]
                      - ((globalIdx_XY.x - disparity >= 0) ?
                          d_imgInright[id + chOffset - disparity] : 0.f));
            }
          d_imgOutOld[disparity][id] = 0.f;
          d_imgOutFit[disparity][id] = 0.f;
          d_f[(size_t) disparity * imgDims.x * imgDims.y + id] = init_value
              * mu;
        }
      //Disparity Boundary Cdn
      d_imgOutOld[0][id] = 1.f;
    }
}

__global__ void
initialize_tm(float *d_f, uint32_t nc, dim3 imgDims, float **d_imgOutOld,
    float **d_imgOutFit, uint32_t disparities, uint32_t mu)
{

  dim3 globalIdx_XY = globalIdx_Dim2();
  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      float init_value;
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          init_value = 0.f;
          // for all channels
          for (uint32_t ch_i = 0; ch_i < nc; ch_i++)
            {
              init_value += fabsf(
                  tex2D(texRefleftImage, globalIdx_XY.x + 0.5f,
                      globalIdx_XY.y + (size_t) ch_i * imgDims.y + 0.5f)
                      - tex2D(texRefrightImage,
                          globalIdx_XY.x + 0.5f - disparity,
                          globalIdx_XY.y + (size_t) ch_i * imgDims.y + 0.5f));
            }
          d_imgOutOld[disparity][id] = 0.f;
          d_imgOutFit[disparity][id] = 0.f;
          d_f[(size_t) disparity * imgDims.x * imgDims.y + id] = init_value
              * mu;
        }
      //Disparity Boundary Cdn
      d_imgOutOld[0][id] = 1.f;
    }
}

__global__ void
initialize_dual(float **dptr_phiX, float **dptr_phiY, float **dptr_phiZ,
    uint32_t disparities, dim3 imgDims)
{

  dim3 globalIdx_XY = globalIdx_Dim2();

  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);

      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          dptr_phiX[disparity][id] = 0.f;
          dptr_phiY[disparity][id] = 0.f;
          dptr_phiZ[disparity][id] = 0.f;
        }
    }
}

__global__ void
regularizer_update(float **dptr_phiX, float **dptr_phiY, float **dptr_phiZ,
    float **dptr_imgOutFit, float *d_f, float sigma, dim3 imgDims,
    uint32_t disparities)
{

  dim3 globalIdx_XY = globalIdx_Dim2();

  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      float d_primalFit, dphiNorm, dphiX, dphiY, dphiZ;
      size_t neighX, neighY;
      // for all channels
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          // get linear ids of neighbours of offset +1 in x, y and z dir
          neighX = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(1, 0, 0));
          neighY = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(0, 1, 0));

          d_primalFit = dptr_imgOutFit[disparity][id];

          dphiX = dptr_phiX[disparity][id]
              + sigma
                  * (((globalIdx_XY.x + 1) < imgDims.x) ?
                      (dptr_imgOutFit[disparity][neighX] - d_primalFit) : 0);
          dphiY = dptr_phiY[disparity][id]
              + sigma
                  * (((globalIdx_XY.y + 1) < imgDims.y) ?
                      (dptr_imgOutFit[disparity][neighY] - d_primalFit) : 0);
          dphiZ = dptr_phiZ[disparity][id]
              + sigma
                  * ((disparity + 1) < disparities ?
                      (dptr_imgOutFit[disparity + 1][id] - d_primalFit) : 0);

          //Projection and maintaining the constraints
          dphiNorm = powf(dphiX, 2) + powf(dphiY, 2);
          dphiNorm = fmaxf(1.f, sqrtf(dphiNorm));
          dptr_phiX[disparity][id] = dphiX / dphiNorm;
          dptr_phiY[disparity][id] = dphiY / dphiNorm;

          //Forward translation
          dphiZ += d_f[(size_t) disparity * imgDims.x * imgDims.y + id];
          // Total Variation Term + projection constraint
          dphiZ = fmaxf(0.f, dphiZ);
          //Backward translation
          dptr_phiZ[disparity][id] = dphiZ
              - d_f[(size_t) disparity * imgDims.x * imgDims.y + id];
        }
    }
}

__global__ void
regularizer_update_tm(float **dptr_phiX, float **dptr_phiY, float **dptr_phiZ,
    float **dptr_imgOutFit, float sigma, dim3 imgDims, uint32_t disparities)
{

  dim3 globalIdx_XY = globalIdx_Dim2();

  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      float d_primalFit, dphiNorm, dphiZnorm, dphiX, dphiY, dphiZ;
      size_t neighX, neighY;
      // for all channels
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          // get linear ids of neighbours of offset +1 in x, y and z dir
          neighX = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(1, 0, 0));
          neighY = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(0, 1, 0));

          d_primalFit = dptr_imgOutFit[disparity][id];

          dphiX = dptr_phiX[disparity][id]
              + sigma
                  * (((globalIdx_XY.x + 1) < imgDims.x) ?
                      (dptr_imgOutFit[disparity][neighX] - d_primalFit) : 0);
          dphiY = dptr_phiY[disparity][id]
              + sigma
                  * (((globalIdx_XY.y + 1) < imgDims.y) ?
                      (dptr_imgOutFit[disparity][neighY] - d_primalFit) : 0);
          dphiZ = dptr_phiZ[disparity][id]
              + sigma
                  * ((disparity + 1) < disparities ?
                      (dptr_imgOutFit[disparity + 1][id] - d_primalFit) : 0);

          //Projection and maintaining the constraints
          dphiNorm = powf(dphiX, 2) + powf(dphiY, 2);
          dphiNorm = fmaxf(1.f, sqrtf(dphiNorm));
          dptr_phiX[disparity][id] = dphiX / dphiNorm;
          dptr_phiY[disparity][id] = dphiY / dphiNorm;

          //Forward translation
          dphiZ += tex3D(texRefDataTerm, globalIdx_XY.x, globalIdx_XY.y,
              disparity);
          //Total Variation Term + projection constraint
          dphiZ = fmaxf(0.f, dphiZ);
          //Backward translation
          dptr_phiZ[disparity][id] = dphiZ
              - tex3D(texRefDataTerm, globalIdx_XY.x, globalIdx_XY.y,
                  disparity);
        }
    }
}

__global__ void
variational_update(float **dptr_imgOutOld, float **dptr_phiX, float **dptr_phiY,
    float **dptr_phiZ, float **dptr_imgOutFit, float tau, dim3 imgDims,
    uint32_t disparities)
{

  dim3 globalIdx_XY = globalIdx_Dim2();

  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      // get linear index
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      float div_phi, d_PixelNew, d_PixelPrev;
      size_t neighX, neighY;
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          div_phi = 0.f;
          neighX = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(-1, 0, 0));
          neighY = linearize_neighbour_globalIdx(globalIdx_XY, imgDims,
              make_int3(0, -1, 0));

          div_phi += (
              (globalIdx_XY.x + 1) < imgDims.x ? dptr_phiX[disparity][id] : 0)
              - (globalIdx_XY.x > 0 ? dptr_phiX[disparity][neighX] : 0);

          div_phi += (
              (globalIdx_XY.y + 1) < imgDims.y ? dptr_phiY[disparity][id] : 0)
              - (globalIdx_XY.y > 0 ? dptr_phiY[disparity][neighY] : 0);

          div_phi += (
              (disparity + 1 < disparities) ? dptr_phiZ[disparity][id] : 0)
              - ((disparity > 0) ? dptr_phiZ[disparity - 1][id] : 0);

          d_PixelPrev = dptr_imgOutOld[disparity][id];
          d_PixelNew = d_PixelPrev + tau * div_phi;
          // Clipping the depthOutput  to range [0,1]
          d_PixelNew =
              (disparity == 0) ?
                  1.f :
                  (float) (disparity != disparities - 1)
                      * fminf(1.f, fmaxf(0.f, d_PixelNew));

          //Updating the fitted image
          dptr_imgOutFit[disparity][id] = 2.f * d_PixelNew - d_PixelPrev;
          dptr_imgOutOld[disparity][id] = d_PixelNew;
        }
    }
}

__global__ void
layers_summation(float *d_imgOut, float **dptr_imgOutOld, uint32_t disparities,
    dim3 imgDims)
{
  dim3 globalIdx_XY = globalIdx_Dim2();
  if (globalIdx_XY.x < imgDims.x && globalIdx_XY.y < imgDims.y)
    {
      size_t id = linearize_globalIdx(globalIdx_XY, imgDims);
      d_imgOut[id] = 0.f;
      for (uint32_t disparity = 0; disparity < disparities; disparity++)
        {
          d_imgOut[id] += (dptr_imgOutOld[disparity][id] > 0.5f ? 1.f : 0.f);
        }
    }
}

