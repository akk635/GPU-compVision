// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ###
// ###

#include <aux.h>
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA
#include "stereo_projection.h"
#include "disparity_stereo.h"

int
main(int argc, char **argv)
{
  // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
  // This happens on the very first call to a CUDA function, and takes some time (around half a second)
  // We will do it right here, so that the run time measurements are accurate
  hipDeviceSynchronize();
  CUDA_CHECK;

  // Reading command line parameters:
  // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
  // If "-param" is not specified, the value of "var" remains unchanged
  //
  // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
  // input image - left and right
  string imageLeft = "", imageRight = "";
  bool ret = getParam("i_left", imageLeft, argc, argv)
      && getParam("i_right", imageRight, argc, argv);
  if (!ret)
    cerr << "ERROR: one or more image(s) not specified" << endl;
  if (argc <= 1)
    {
      cout << "Usage: " << argv[0]
          << " -i_left <image> -i_right <image> [-repeats <repeats>] [-gray]"
          << endl;
      return 1;
    }
#endif

  // suppress all outputs by program except file writes and errors
  bool suppressOut = false;
  getParam("suppress_out", suppressOut, argc, argv);
  cout << "Suppress Out: " << suppressOut << endl;

  // number of computation repetitions to get a better run time measurement
  int repeats = 1;
  getParam("repeats", repeats, argc, argv);

  // load the input image as grayscale if "-gray" is specifed
  bool gray = false;
  getParam("gray", gray, argc, argv);

  // ### Define your own parameters here as needed
  // get MU
  float MU;
  bool retVal = getParam("mu", MU, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no MU specified" << endl;
      cout << "Usage: " << argv[0] << " -mu <value> " << endl;
      return 1;
    }

  // get sigma
  float SIGMA;
  retVal = getParam("sigma", SIGMA, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no SIGMA specified" << endl;
      cout << "Usage: " << argv[0] << " -sigma <value>" << endl;
      return 1;
    }

  // get TAU
  float TAU;
  retVal = getParam("tau", TAU, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no TAU specified" << endl;
      cout << "Usage: " << argv[0] << " -tau <value>" << endl;
      return 1;
    }

  // get discretization
  uint32_t nt;
  retVal = getParam("nt", nt, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no discretization specified" << endl;
      cout << "Usage: " << argv[0] << " -nt <value>" << endl;
      return 1;
    }

  // get steps
  uint32_t steps;
  retVal = getParam("steps", steps, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no step specified" << endl;
      cout << "Usage: " << argv[0] << " -steps <value>" << endl;
      return 1;
    }

  // get implementation type
  int impl;
  retVal = getParam("impl", impl, argc, argv);
  if (!retVal)
    {
      cerr << "ERROR: no implementation specified" << endl;
      cout << "Usage: " << argv[0]
          << " -impl <value> {0:'Global Memory', 1:'Texture', 2:'Pitch', 3:'Shared Memory', 4:'2D Grid'}"
          << endl;
      return 1;
    }

  if (!suppressOut)
    {
      // output parameters
      cout << "repeats: " << repeats << endl;
      cout << "gray: " << gray << endl;
      cout << "MU: " << MU << endl;
      cout << "SIGMA: " << SIGMA << endl;
      cout << "TAU: " << TAU << endl;
      cout << "nt: " << nt << endl;
      cout << "Steps: " << steps << endl;
      cout << "Impl: " << impl << endl;

    }

  // Init camera / Load input image
#ifdef CAMERA

  // Init camera
  cv::VideoCapture camera(0);
  if(!camera.isOpened())
    { cerr << "ERROR: Could not open camera" << endl; return 1;}
  int camW = 640;
  int camH = 480;
  camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
  // read in first frame to get the dimensions
  cv::Mat mInLeft;
  camera >> mInLeft;

#else

  // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
  cv::Mat mInLeft = cv::imread(imageLeft.c_str(),
      (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
  // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
  cv::Mat mInRight = cv::imread(imageRight.c_str(),
      (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
  // check if both images are loaded
  if (mInLeft.data == NULL || mInRight.data == NULL)
    {
      cerr << "ERROR: Could not load one or more image(s) specified" << endl;
      return 1;
    }

#endif

  // convert to float representation (opencv loads image values as single bytes by default)
  mInLeft.convertTo(mInLeft, CV_32F);
  mInRight.convertTo(mInRight, CV_32F);
  // convert range of each channel to [0,1] (opencv default is [0,255])
  mInLeft /= 255.f;
  mInRight /= 255.f;

  // get image dimensions and both must be same
  if (mInLeft.cols != mInRight.cols || mInLeft.rows != mInRight.rows)
    {
      cerr << "ERROR: Image dimensions don't match!" << endl;
      return 1;
    }
  int w = mInLeft.cols;         // width
  int h = mInLeft.rows;         // height
  int nc = mInLeft.channels();  // number of channels
  if (!suppressOut)
    cout << "image dimensions: " << w << " x " << h << endl;

  //cv::Mat mOut(h,w,mInLeft.type());  // mOut will have the same number of channels as the input image, nc layers
  //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
  // ### Define your own output images here as needed
  cv::Mat mOutDepth(h, w, CV_32FC1);    // mOut will be a color image, 1 layers

  // Allocate arrays
  // input/output image height: h
  // input image number of channels: nc
  // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)
  // allocate raw input image array
  float *imgInLeft = new float[(size_t) w * h * nc];
  float *imgInRight = new float[(size_t) w * h * nc];
  // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
  float *imgOutDepth = new float[(size_t) w * h * mOutDepth.channels()];

  // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
  // Read a camera image frame every 30 milliseconds:
  // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
  // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
  while (cv::waitKey(30) < 0)
    {
      // Get camera image
      camera >> mInLeft;
      // convert to float representation (opencv loads image values as single bytes by default)
      mInLeft.convertTo(mInLeft,CV_32F);
      // convert range of each channel to [0,1] (opencv default is [0,255])
      mInLeft /= 255.f;
#endif

  // Init raw input image array
  // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
  // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
  // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
  convert_mat_to_layered(imgInLeft, mInLeft);
  convert_mat_to_layered(imgInRight, mInRight);

  Timer timer;
  timer.start();

  // GPU version based on implementation type selected
  switch (impl)
    {
  case 0:
    stereo_projection_PD(imgInLeft, imgInRight, imgOutDepth, dim3(w, h, 0), nc,
        dim3(w, h, nt), steps, MU, SIGMA, TAU);
    break;
  case 1:
    stereo_projection_PD_tex(imgInLeft, imgInRight, imgOutDepth, dim3(w, h, 0),
        nc, dim3(w, h, nt), steps, MU, SIGMA, TAU);
    break;
  case 2:
    stereo_projection_PD_pitch(imgInLeft, imgInRight, imgOutDepth,
        dim3(w, h, 0), nc, dim3(w, h, nt), steps, MU, SIGMA, TAU);
    break;
  case 3:
    stereo_projection_PD_sm(imgInLeft, imgInRight, imgOutDepth, dim3(w, h, 0),
        nc, dim3(w, h, nt), steps, MU, SIGMA, TAU);
    break;
  case 4:
    disparity_computation_caller(imgInLeft, imgInRight, imgOutDepth,
        dim3(w, h, 0), nc, SIGMA, TAU, steps, MU, nt); break;
  case 5:
    disparity_computation_caller_tm(imgInLeft, imgInRight, imgOutDepth,
        dim3(w, h, 0), nc, SIGMA, TAU, steps, MU, nt); break;
  default:
    break;
    }

  timer.end();
  float t = timer.get();  // elapsed time in seconds
  if (!suppressOut)
    cout << "time: " << t * 1000 << " ms" << endl;

  // for output calculation convert back to interleaved open cv and normalize
  convert_layered_to_mat(mOutDepth, imgOutDepth);
  double minVal, maxVal;
  minMaxLoc(mOutDepth, &minVal, &maxVal);
  mOutDepth /= maxVal;

  // show input image
  // show at position (x_from_left=100,y_from_above=100)
  if (!suppressOut)
    showImage("Input Left", mInLeft, 100, 100);
  // show at position (x_from_left=100,y_from_above=100)
  if (!suppressOut)
    showImage("Input Right", mInRight, 100 + w + 40, 100);

  // ### Display your own output images here as needed
  if (!suppressOut)
    showImage("Depth Mapping", mOutDepth, 100 + 2 * w + 40, 100);

#ifdef CAMERA
  // end of camera loop
}
#else
  // wait for key inputs
  cv::waitKey(0);
#endif

  // save input and result
  cv::imwrite("images/out/depth_map.png", mOutDepth * 255.f);

  // free allocated arrays
  delete[] imgInLeft;
  delete[] imgInRight;
  delete[] imgOutDepth;

  // close all opencv windows
  cvDestroyAllWindows();
  return 0;
}
