#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###



// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ###
// ###


#include <aux.h>
#include <iostream>
#include <math.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

#include "inpainting_gradient_descent.h"



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();
    CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    




    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;


    //cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut(h, w, mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mMask(h, w, CV_32FC1);  // mOut will have the same number of channels as the input image, nc layers
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w * h * nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w * h * mOut.channels()];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgMask = new float[(size_t)w * h * mMask.channels()];



    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    // get tau
    float TAU;
    bool retVal = getParam("tau", TAU, argc, argv);
    if (!retVal) {
        cerr << "ERROR: no TAU specified" << endl;
        cout << "Usage: " << argv[0] << " -tau <value> " << endl; return 1;
    }

    // get epsilon
    float EPSILON;
    retVal = getParam("epsilon", EPSILON, argc, argv);
    if (!retVal) {
        cerr << "ERROR: no EPSILON specified" << endl;
    cout << "Usage: " << argv[0] << " -epsilon <value>" << endl; return 1;
    }

    // get steps
    uint32_t steps;
    retVal = getParam("steps", steps, argc, argv);
    if (!retVal) {
        cerr << "ERROR: no step specified" << endl;
        cout << "Usage: " << argv[0] << " -steps <value>" << endl; return 1;
    }

    // get diffusivity type
    const uint32_t MAX_DIFF_TYPE = 2;
    uint32_t diffType;
    retVal = getParam("diffusivity_type", diffType, argc, argv);
    if (!retVal || diffType > MAX_DIFF_TYPE) {
        cerr << "ERROR: no step specified or invalid value given" << endl;
        cout << "Usage: " << argv[0] << " -diffusivity_type <key-value> from {0: 1/max(s, e), 1: 1, 2: exp(-s^2/e)/e}" << endl; return 1;
    }

    // output parameters
    cout << "TAU: " << TAU << endl;
    cout << "EPSILON: " << EPSILON << endl;
    cout << "Steps: " << steps << endl;
    cout << "Diffusivity type: " << diffType << endl;
   
    Timer timer; timer.start();

    // GPU version
    inpainting_gradient_descent(imgIn, imgMask, imgOut, dim3(w, h, 0), nc, make_float3(0, 1, 0), make_float3(0.5, 0.5, 0.5), TAU, EPSILON, steps, diffType);

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // ### Display your own output images here as needed
    // show output image: first convert to interleaved opencv format from the layered raw array    
    convert_layered_to_mat(mOut, imgOut);
    showImage("Inpainted", mOut, 100+w+40, 100);

    convert_layered_to_mat(mMask, imgMask);
    showImage("Mask", mMask, 100+w+140, 100);
    


#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_result.png",mOut * 255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



