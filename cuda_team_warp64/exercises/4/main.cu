#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###



// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### Edited by: Shoubhik Debnath (p052)
// ###
// ###


#include "aux.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA

// threshold kernel
__global__ void threshold_image(float *d_a, float *d_b, size_t width, size_t height, size_t nc) {
    // get thread id
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    float threshold = 0.6;
    float value = 0;
    
     // only threads inside array range compute
    if(id < width*height)
    {    
        for(size_t i = 0; i < nc; i++)
        {
            value += d_a[(width*height*i)+id];
        }
        d_b[id] = (value/nc) > threshold ? 1:0;
    }
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    




    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    //cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    Timer timer; timer.start();
    // CPU version
    /*int r_start = 0;
    int g_start = w*h;
    int b_start = 2*w*h;
    
    float value = 0;
    float threshold = 0.6;
    for(size_t i = 0; i < w*h; i++)
    {
        value = (imgIn[r_start+i] + imgIn[g_start+i] + imgIn[b_start+i])/3;        
        if(value > threshold)
            imgOut[i] = 1;
        else 
            imgOut[i] = 0;
    }*/
    
    // GPU version
    
    size_t n = w*h;
    float *h_a = imgIn;
    float *h_b = imgOut;
    
    // define block and grid sizes - 1D assumed
    // setting a block of 128 threads
    dim3 block = dim3(128, 1, 1);
    dim3 grid = dim3((n + block.x - 1) / block.x, 1, 1);
    
    // alloc GPU memeory and copy data
    float *d_a;
    hipMalloc((void **) &d_a, n * nc * sizeof(float));
    hipMemcpy(d_a, h_a, n * nc * sizeof(float), hipMemcpyHostToDevice);
    
    float *d_b;
    hipMalloc((void **) &d_b, n * sizeof(float));
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);
    
    // call kernel
    threshold_image<<<grid, block>>>(d_a, d_b, (size_t)w, (size_t)h, (size_t)nc);
    
    // wait for kernel call to finish
    hipDeviceSynchronize();
    
    // check for error
    hipGetLastError();
    
    // copy back data
    hipMemcpy(h_b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);
    
    
    // free GPU array
    hipFree(d_a);
    hipFree(d_b);
    
    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



